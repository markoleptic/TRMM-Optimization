// -*- mode: c++ -*-
/*
  This is the baseline implementation of a Triangular Matrix Times Matrix
  Multiplication  (TRMM)

  C = AB, where
  A is an MxM lower triangular (A_{i,p} = 0 if p > i) Matrix. It is indexed by i0 and p0
  B is an MxN matrix. It is indexed by p0 and j0.
  C is an MxN matrix. It is indexed by i0 and j0.


  Parameters:

  m0 > 0: dimension
  n0 > 0: dimension



  float* A_sequential: pointer to original A matrix data
  float* A_distributed: pointer to the input data that you have distributed across
  the system

  float* C_sequential:  pointer to original output data
  float* C_distributed: pointer to the output data that you have distributed across
  the system

  float* B_sequential:  pointer to original weights data
  float* B_distributed: pointer to the weights data that you have distributed across
  the system

  Functions:

  DISTRIBUTED_ALLOCATE_NAME(...): Allocate the distributed buffers.
  DISTRIBUTE_DATA_NAME(...): takes the sequential data and distributes it across the system.
  COMPUTE_NAME(...): Performs the stencil computation.
  COLLECT_DATA_NAME(...): Collect the distributed output and combine it back to the sequential
  one for testing.
  DISTRIBUTED_FREE_NAME(...): Free the distributed buffers that were allocated


  - richard.m.veras@ou.edu

*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans)                                                                                                 \
	{                                                                                                              \
		gpuAssert((ans), __FILE__, __LINE__);                                                                  \
	}
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort)
			exit(code);
	}
}

extern "C" void compute_device(int m0, int n0, float *A_distributed, float *B_distributed, float *C_distributed);

extern "C" void allocate_device(int m0, int n0, float **A_device, float **B_device, float **C_device);

extern "C" void free_device(int m0, int n0, float *A_device, float *B_device, float *C_device);

extern "C" void collect_data_from_device(int m0, int n0, float *C_device, float *C_distributed);

extern "C" void distribute_data_to_device(int m0, int n0, float *A_distributed, float *B_distributed, float *A_device,
					  float *B_device);

/* This is the GPU kernel. */
__global__ void cuda_trmm(int m0, int n0, float *A_device, float *B_device, float *C_device)
{
	/*
	  student_todo: this is where the majority of the work will happen.
	 */

	/*
	  Using the convention that row_stride (rs) is the step size you take going down a row,
	  column stride (cs) is the step size going down the column.
	*/
	// A is column major
	int rs_A = m0;
	int cs_A = 1;

	// B is column major
	int rs_B = m0;
	int cs_B = 1;

	// C is column major
	int rs_C = m0;
	int cs_C = 1;

	int id = blockDim.x * blockIdx.x + threadIdx.x;

	for (int j0 = id; j0 < n0; j0 += blockDim.x * gridDim.x)
	{
		for (int p0 = 0; p0 < m0; ++p0)
		{
			float B_pj = B_device[p0 * cs_B + j0 * rs_B];
			for (int i0 = 0; i0 < j0; ++i0)
			{
				float A_ip = A_device[i0 * cs_A + p0 * rs_A];
				C_device[i0 * cs_C + j0 * rs_C] += A_ip * B_pj;
			}
		}
	}
}

void allocate_device(int m0, int n0, float **A_device, float **B_device, float **C_device)
{
	int bytes_A = m0 * m0 * sizeof(float);
	int bytes_B = m0 * n0 * sizeof(float);
	int bytes_C = m0 * n0 * sizeof(float);

	/**/

	// printf("GPU Allocate: ");
	gpuErrchk(hipMalloc(A_device, bytes_A));
	gpuErrchk(hipMalloc(B_device, bytes_B));
	gpuErrchk(hipMalloc(C_device, bytes_C));
	// printf("Done\n");
}

void distribute_data_to_device(int m0, int n0, float *A_distributed, float *B_distributed, float *A_device,
			       float *B_device)
{
	int bytes_A = m0 * m0 * sizeof(float);
	int bytes_B = m0 * n0 * sizeof(float);

	// printf("GPU Distribute: ");

	/* student_todo: you can modify this code if you want to lay out the data in
			 the gpu in a particular way. */

	gpuErrchk(hipMemcpy(A_device, A_distributed, bytes_A, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(B_device, B_distributed, bytes_B, hipMemcpyHostToDevice));
	// printf("Done\n");
}

void collect_data_from_device(int m0, int n0, float *C_device, float *C_distributed)
{
	int bytes_C = m0 * n0 * sizeof(float);

	// printf("GPU Collect: ");

	/* student_todo: you can modify this code if you want to lay out the data in
			 the gpu in a particular way. */

	gpuErrchk(hipMemcpy(C_distributed, C_device, bytes_C, hipMemcpyDeviceToHost));
	// printf("Done\n");
}

void free_device(int m0, int n0, float *A_device, float *B_device, float *C_device)
{
	// Free GPU memory
	// printf("GPU Free: ");
	gpuErrchk(hipFree(A_device));
	gpuErrchk(hipFree(B_device));
	gpuErrchk(hipFree(C_device));
	// printf("Done\n");
}

void compute_device(int m0, int n0, float *A_distributed, float *B_distributed, float *C_distributed)

{

	// printf("GPU Compute: ");

	// student_todo: you will sweep through these knobs
	// These are knobs you can tune

    int threads_per_block = 1;
	int blocks_per_grid = (m0 * n0) / threads_per_block;

	// Run the kernel
	cuda_trmm<<<blocks_per_grid, threads_per_block>>>(m0, n0, A_distributed, B_distributed, C_distributed);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	//  printf("Done\n");
}
